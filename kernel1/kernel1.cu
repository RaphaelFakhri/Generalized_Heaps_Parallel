#include "hip/hip_runtime.h"
#include "common.h"



__global__ void initializeHeap(Heap *heap, Partial_Buffer *partialBuffer) {
    
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if(index == 0) {
        // Only use one thread to set the attributes.
        heap -> itemCount = 0;
        partialBuffer -> itemCount = 0;
    }

    // Set Nodes.
    if (index < HEAP_CAPACITY) {
        heap -> arr[index] = INT_MAX;
    }
    // Set Partial Buffer.
    if (index < PARTIAL_BUFFER_CAPACITY) {
        partialBuffer -> arr[index] = INT_MAX;   
    }
}

// First Optimization: Use parallel sorting, this implementation is based on bitonic sort, not radix sort which we had envisioned.
// Performance improvements from using multiple threads, but limited to using power of 2 elements.
// Time complexity: O(log^2(n)) for n elements.
// Space complexity: O(n).
__device__ void sortBitonic(int *arr, int itemCount) {
    int threadId = threadIdx.x;
    int maximum = 0, minimum = 0, otherId = 0, i = 2, j = 2;

    int batchNumber = threadId >> 1; // Parity of batchNumber determines which threads are active during the cycle (half).

    // outer loop: controls size of sorting array. As it increases, we obtain a bitonic sequence and then a sorted array.
    for (i = 2; i <= itemCount ; i <<= 1, batchNumber >>= 1) {
        // middle loop: performs bitonic swaps for chunks of bitonic sequences.
        for (j = i; j >= 2 ; j >>= 1) {
            int lookAheadSteps = j >> 1;
            if ((threadId % j < lookAheadSteps) && (threadId < itemCount)) 
            {
                otherId = threadId + lookAheadSteps;
                minimum = min(arr[threadId], arr[otherId]);
                maximum = max(arr[threadId], arr[otherId]); 
                if (batchNumber & 1) {
                    arr[threadId] = maximum;
                    arr[otherId] = minimum;
                }
                else {
                    arr[threadId] = minimum;
                    arr[otherId] = maximum;
                }
            }
            __syncthreads();
        }
    }
}

// Parallel array copy algorithm to utilize multiple threads. 
// Not considered an optimization because it is fairly straightforward.
// Time Complexity: O(n)
__device__ void copyArray(int *arr1, int startIndex1, int *arr2, int startIndex2, int elementCount) {
    int threadId = threadIdx.x;
    if (threadId < elementCount) {
        arr2[startIndex2 + threadId] = arr1[startIndex1 + threadId];
    }
    __syncthreads();
}


// Parallel version of the memset function, needed to reset the values of all keys inside a node at once (for example when deleting a node).
// Not considered an optimization because it is fairly straightforward.
// Time Complexity: O(n)

__device__ void memsetArray(int *arr, int arrStartIndex, int val, int elementCount) {
    // sets values of arr to val between given indices of arr
    int threadId = threadIdx.x;
    if (threadId < elementCount) {
        arr[arrStartIndex + threadId] = val;
    }
    __syncthreads();
}

// Bit reversal is useful to create a root to target node path.
// Time Complexity: O(n).
__device__ int reverseBits(int n, int level) {

    if (n <= 4) 
        return n;

    int ans = 1 << (level--);
    while(n != 1) {
        ans += ((n & 1) << (level--));
        n >>= 1;
    }
    return ans;
}

// Lock Management functions, instead of locking the entire heap, we lock individual nodes.
// Limitation: Since we always lock the root node and it is always accessed, we are effectively locking the entire heap.

// Acquire a lock if available.
__device__ void acquireLock(int *lock, int initialState, int finalState) {
    while (atomicCAS(lock, initialState, finalState) != initialState);
}

// Make node available.
__device__ void releaseLockAtomic(int *lock, int initialState, int finalState) {
    atomicCAS(lock, initialState, finalState);
}

// Binary search algorithm to find the index of the smallest element larger than the element searched in arr1 (to maintain heap property).
// Time Complexity: O(log(n)).

__device__ int binarySearch(int *arr1, int high, int search, bool considerEquality) {
    // Boundary conditions
    if(high == 0) return 0;
    int low = 0, mid = 0;
    int ans = high;
    while (low <= high)
    {
        mid = (low + high) >> 1;
        if (arr1[mid] >= search and considerEquality) { // Higher or Equal? LeftChild half
            ans = mid;
            high = mid - 1;
        }
        else if (arr1[mid] > search) { // Strictly Higher? LeftChild half
            ans = mid;
            high = mid - 1;
        }
        else { // Lower? RightChild half
            low = mid + 1;
        }
    }
    return ans;
}

// Merge and sort two arrays in parallel, used to merge nodes from root to target node to preserve heap property.
// Not considered an optimization because it is quite necessary and clearly explained in the paper.
// Time Complexity: O(n log(n)) (due to the binary search), where n is the larger of the two arrays.
// Space Complexity: O(n).

__device__ void mergeAndSortArrays(int *arr1, int idx1, int *arr2, int idx2, int *mergedArr) {

    __syncthreads();
    // Special cases
    if(idx1 == 0) {
        copyArray(arr2, 0, mergedArr, 0, idx2);
    }
    else if(idx2 == 0) {
        copyArray(arr1, 0, mergedArr, 0, idx1);
    }
    
    else if(arr1[idx1 - 1] <= arr2[0]) {
        copyArray(arr1, 0, mergedArr, 0, idx1);
        copyArray(arr2, 0, mergedArr, idx1, idx2);
    }

    else if(arr2[idx2 - 1] <= arr1[0]) {\
        copyArray(arr2, 0, mergedArr, 0, idx2);
        copyArray(arr1, 0, mergedArr, idx2, idx1);
    }
    // General case
    else {

        int threadId = threadIdx.x;
        if (threadId < idx1) {
            int x = binarySearch(arr2, idx2, arr1[threadId], 1);
            mergedArr[threadId + x] = arr1[threadId];
        }

        if (threadId < idx2) {
            int x = binarySearch(arr1, idx1, arr2[threadId], 0);
            mergedArr[threadId + x] = arr2[threadId];
        }
    }
    __syncthreads();
    
}

__global__ void topDownInsertion(int *itemsToInsert, int itemCount, int *heapLocks, Partial_Buffer *partialBuffer, Heap *heap, int *mergedResult) {
    int threadId = threadIdx.x;

    int *insertionItems = itemsToInsert;
    sortBitonic(insertionItems, itemCount);

    if (threadId == MASTER_THREAD){
        acquireLock(&heapLocks[ROOT_NODE_IDX], AVAILABLE, INUSE);
    }
    __syncthreads();

    int combinedItemCount = partialBuffer->itemCount + itemCount;

    mergeAndSortArrays(insertionItems, itemCount, partialBuffer->arr, partialBuffer->itemCount, mergedResult);

    // Check if the combined item count exceeds the batch size
    if (combinedItemCount >= BATCH_SIZE) {
        copyArray(mergedResult, 0, insertionItems, 0, BATCH_SIZE);
        copyArray(mergedResult, BATCH_SIZE , partialBuffer->arr, 0, combinedItemCount - BATCH_SIZE);
        __threadfence();

        if (threadId == MASTER_THREAD)
            atomicExch(&(partialBuffer->itemCount), combinedItemCount - BATCH_SIZE);
        __syncthreads();
    } else {
        if (heap->itemCount == 0) {
            copyArray(mergedResult, 0, partialBuffer->arr, 0, combinedItemCount);
            __threadfence();
        } else {
            copyArray(mergedResult, 0, heap->arr, ROOT_NODE_IDX * BATCH_SIZE, BATCH_SIZE);
            __syncthreads();

            mergeAndSortArrays(heap->arr, ROOT_NODE_IDX * BATCH_SIZE, insertionItems, BATCH_SIZE, mergedResult);

            copyArray(mergedResult, 0, heap->arr, ROOT_NODE_IDX * BATCH_SIZE, BATCH_SIZE);
            __threadfence();

            copyArray(mergedResult, BATCH_SIZE, partialBuffer->arr, 0, combinedItemCount);
            __threadfence();
        }
        if (threadId == MASTER_THREAD)
            partialBuffer->itemCount = combinedItemCount;

        __syncthreads();

        if (threadId == MASTER_THREAD)
            releaseLockAtomic(&heapLocks[ROOT_NODE_IDX], INUSE, AVAILABLE);
        return;
    }

    // Update the heap item count
    if (threadId == MASTER_THREAD)
        atomicAdd(&(heap->itemCount), 1);
    __syncthreads();

    // Calculate the target node and its level in the heap
    int targetNode = heap->itemCount, level = -1;
    int tempTargetNode = targetNode;
    while(tempTargetNode) {
        level++;
        tempTargetNode >>= 1;
    }

    // Reverse the bits to find the correct position in the heap
    targetNode = reverseBits(targetNode, level);
    
    // Lock management to avoid deadlocks and ensure thread synchronization
    if (targetNode != ROOT_NODE_IDX) {
        if (threadId == MASTER_THREAD) {
            acquireLock(&heapLocks[targetNode], AVAILABLE, INUSE);
        }
        __syncthreads();
    }

    int low = 0, currentNode = ROOT_NODE_IDX;
    while (currentNode != targetNode) {
        low = currentNode * BATCH_SIZE;
       
        copyArray(heap->arr, low, mergedResult, 0, BATCH_SIZE);

        mergeAndSortArrays(mergedResult, BATCH_SIZE, insertionItems, BATCH_SIZE, mergedResult);

        copyArray(mergedResult, 0, heap->arr, low, BATCH_SIZE);
        __threadfence();

        copyArray(mergedResult, BATCH_SIZE, insertionItems, 0, BATCH_SIZE);

        currentNode = targetNode >> (--level);

        if (threadId == MASTER_THREAD) {
            if (currentNode != targetNode) {
                acquireLock(&heapLocks[currentNode], AVAILABLE, INUSE);
            }
            releaseLockAtomic(&heapLocks[currentNode >> 1], INUSE, AVAILABLE);
        }
        __syncthreads();
    }

    copyArray(insertionItems, 0, heap->arr, targetNode * BATCH_SIZE, BATCH_SIZE);
    __threadfence();
    if (threadId == MASTER_THREAD) {
        releaseLockAtomic(&heapLocks[targetNode], INUSE, AVAILABLE);
    }
    __syncthreads();
}


__global__ void topDownDeletion(int* deletedItems, int* heapLocks, Partial_Buffer* partialBuffer, Heap* heap, int* array1, int* array2, int* array3, int* mergedResult) {
    int threadId = threadIdx.x;

    if (threadId == MASTER_THREAD) {
        acquireLock(&heapLocks[ROOT_NODE_IDX], AVAILABLE, INUSE);
    }
    __syncthreads();

    if (heap->itemCount == 0) {
        if (partialBuffer->itemCount != 0) {
            copyArray(partialBuffer->arr, 0, deletedItems, 0, partialBuffer->itemCount);
            __threadfence();
            if (threadId == MASTER_THREAD) {
                atomicExch(&(partialBuffer->itemCount), 0);
            }
            __syncthreads();
        }
        if (threadId == MASTER_THREAD) {
            releaseLockAtomic(&heapLocks[ROOT_NODE_IDX], INUSE, AVAILABLE);
        }
        return;
    }

    copyArray(heap->arr, ROOT_NODE_IDX * BATCH_SIZE, deletedItems, 0, BATCH_SIZE);
    memsetArray(heap->arr, ROOT_NODE_IDX * BATCH_SIZE, INT_MAX, BATCH_SIZE);
    __threadfence();

    int targetNode = heap->itemCount, level = -1;
    int tempTargetNode = targetNode;
    while (tempTargetNode) {
        level++;
        tempTargetNode >>= 1;
    }
    targetNode = reverseBits(targetNode, level);
    __syncthreads();

    if (threadId == MASTER_THREAD) {
        atomicAdd(&(heap->itemCount), -1);
    }
    __syncthreads();

    if (targetNode == 1) {
        if (threadId == MASTER_THREAD) {
            releaseLockAtomic(&heapLocks[ROOT_NODE_IDX], INUSE, AVAILABLE);
        }
        return;
    }

    if (threadId == MASTER_THREAD) {
        acquireLock(&heapLocks[targetNode], AVAILABLE, INUSE);
    }
    __syncthreads();

    copyArray(heap->arr, targetNode * BATCH_SIZE, heap->arr, ROOT_NODE_IDX * BATCH_SIZE, BATCH_SIZE);
    memsetArray(heap->arr, targetNode * BATCH_SIZE, INT_MAX, BATCH_SIZE);
    __threadfence();

    if (threadId == MASTER_THREAD) {
        releaseLockAtomic(&heapLocks[targetNode], INUSE, AVAILABLE);
    }
    __syncthreads();

    copyArray(heap->arr, ROOT_NODE_IDX * BATCH_SIZE, array1, 0, BATCH_SIZE);
    memsetArray(heap->arr, ROOT_NODE_IDX * BATCH_SIZE, INT_MAX, BATCH_SIZE);
    __threadfence();

    copyArray(partialBuffer->arr, 0, array2, 0, partialBuffer->itemCount);
    __syncthreads();

    mergeAndSortArrays(array1, BATCH_SIZE, array2, partialBuffer->itemCount, mergedResult);

    copyArray(mergedResult, BATCH_SIZE, partialBuffer->arr, 0, partialBuffer->itemCount);
    __threadfence();

    copyArray(mergedResult, 0, array1, 0, BATCH_SIZE);

    int leftChild = 0, rightChild = 0, currentNode = 1;
    int largestLeftChild = 0, largestRightChild = 0;

    while (1) {
        if ((currentNode << 1) >= NUMBER_OF_NODES) {
            break;
        }

        leftChild = currentNode << 1;
        rightChild = leftChild + 1;

        if (threadId == MASTER_THREAD) {
            acquireLock(&heapLocks[leftChild], AVAILABLE, INUSE);
            acquireLock(&heapLocks[rightChild], AVAILABLE, INUSE);
        }
        __syncthreads();

        copyArray(heap->arr, leftChild * BATCH_SIZE, array2, 0, BATCH_SIZE);
        memsetArray(heap->arr, leftChild * BATCH_SIZE, INT_MAX, BATCH_SIZE);
        __threadfence();

        copyArray(heap->arr, rightChild * BATCH_SIZE, array3, 0, BATCH_SIZE);
        memsetArray(heap->arr, rightChild * BATCH_SIZE, INT_MAX, BATCH_SIZE);
        __threadfence();

        largestLeftChild = array2[BATCH_SIZE - 1];
        largestRightChild = array3[BATCH_SIZE - 1];

        mergeAndSortArrays(array2, BATCH_SIZE, array3, BATCH_SIZE, mergedResult);

        if (largestLeftChild > largestRightChild) {
            int temp = leftChild;
            leftChild = rightChild;
            rightChild = temp;
        }

        copyArray(mergedResult, BATCH_SIZE, heap->arr, rightChild * BATCH_SIZE, BATCH_SIZE);
        __threadfence();

        if (threadId == MASTER_THREAD) {
            releaseLockAtomic(&heapLocks[rightChild], INUSE, AVAILABLE);
        }
        __syncthreads();

        copyArray(mergedResult, 0, array2, 0, BATCH_SIZE);

        mergeAndSortArrays(array1, BATCH_SIZE, array2, BATCH_SIZE, mergedResult);

        copyArray(mergedResult, 0, heap->arr, currentNode * BATCH_SIZE, BATCH_SIZE);
        __threadfence();

        if (threadId == MASTER_THREAD) {
            releaseLockAtomic(&heapLocks[currentNode], INUSE, AVAILABLE);
        }
        __syncthreads();

        copyArray(mergedResult, BATCH_SIZE, array1, 0, BATCH_SIZE);
        currentNode = leftChild;
        __syncthreads();
    }

    __syncthreads();

    copyArray(array1, 0, heap->arr, currentNode * BATCH_SIZE, BATCH_SIZE);
    __threadfence();

    if (threadId == MASTER_THREAD) {
        releaseLockAtomic(&heapLocks[currentNode], INUSE, AVAILABLE);
    }
    __syncthreads();
}





__host__ void initializeHeap() {
    hipMalloc(&d_partialBuffer, sizeof(Partial_Buffer));
    hipMalloc(&d_Heap, sizeof(Heap)); 
    hipMalloc((void**)&d_heapLock, (1 + NUMBER_OF_NODES) * sizeof(int)) ;

    hipMemsetAsync(d_heapLock, AVAILABLE, (1 + NUMBER_OF_NODES) * sizeof(int) );
    initializeHeap<<<ceil(HEAP_CAPACITY / 1024), 1024, 0>>>(d_Heap, d_partialBuffer);

    hipDeviceSynchronize();

}

// Helper function to insert keys into the heap.
__host__ void insertKeys(int* itemsToInsert, int itemsToInsertSize) {
    if (itemsToInsertSize < 0) {
        return;
    }
    
    int keysInsertionCount = BATCH_SIZE;
    int* d_mergedResult; // Device pointer for mergedResult
    int mergedResultSize = 2 * BATCH_SIZE; // Adjust size as required
    hipMalloc((void**)&d_mergedResult, mergedResultSize * sizeof(int));

    for (int i = 0; i < itemsToInsertSize; i += BATCH_SIZE) {
        keysInsertionCount = min(itemsToInsertSize - i, BATCH_SIZE);
        topDownInsertion<<<1, BLOCK_SIZE>>>(itemsToInsert + i, keysInsertionCount,
                                     d_heapLock, d_partialBuffer, d_Heap, d_mergedResult);
        hipDeviceSynchronize();
    }
    
    hipFree(d_mergedResult);
}

__host__ void deleteKeys(int* deletedItems, int deletedItemsSize) {
    int arraySize = BATCH_SIZE;
    int mergedResultSize = 4 * BATCH_SIZE;  // Adjust as needed, considering operations in the kernel

    int* d_array1, *d_array2, *d_array3, *d_mergedResult;
    hipMalloc((void**)&d_array1, arraySize * sizeof(int));
    hipMalloc((void**)&d_array2, arraySize * sizeof(int));
    hipMalloc((void**)&d_array3, arraySize * sizeof(int));
    hipMalloc((void**)&d_mergedResult, mergedResultSize * sizeof(int));

    for (int i = 0; i < deletedItemsSize; i += BATCH_SIZE) {
        int currentBatchSize = min(deletedItemsSize - i, BATCH_SIZE);
        topDownDeletion<<<1, BLOCK_SIZE>>>(deletedItems + i, d_heapLock, d_partialBuffer, d_Heap, d_array1, d_array2, d_array3, d_mergedResult);
        hipPeekAtLastError();
        hipDeviceSynchronize();
    }

    hipFree(d_array1);
    hipFree(d_array2);
    hipFree(d_array3);
    hipFree(d_mergedResult);
}

// Helper function to insert and delete keys from the heap. This is the third benchmark mentioned by Dr. Izzat
// I acknowledge that using both insertKeys and deleteKeys in this way sort of defeats the purpose of the test
// as it was mentioned that the same method should perform both, through the use of an insertTopDownDeleteTopDown method
// for example, but I was not able to implement such a complex method due to time constraints (although the extensions
// have been generous in this regard). I hope this is acceptable.

__host__ void insertAndDeleteKeys(int* itemsToInsert, int itemsToInsertSize, int* deletedItems, int deletedItemsSize, int interleavingFactor) {
    int keysInsertionCount = BATCH_SIZE;
    int insertCount = 0;
    int deleteCount = 0;

    int arraySize = BATCH_SIZE;
    int mergedResultSize = 4 * BATCH_SIZE;  // Make sure the size matches your requirement

    // Allocate memory for all arrays used in insertion and deletion
    int* d_array1, *d_array2, *d_array3, *d_mergedResultInsert, *d_mergedResultDelete;
    hipMalloc((void**)&d_array1, arraySize * sizeof(int));
    hipMalloc((void**)&d_array2, arraySize * sizeof(int));
    hipMalloc((void**)&d_array3, arraySize * sizeof(int));
    hipMalloc((void**)&d_mergedResultInsert, mergedResultSize * sizeof(int));
    hipMalloc((void**)&d_mergedResultDelete, mergedResultSize * sizeof(int));

    while (insertCount < itemsToInsertSize || deleteCount < deletedItemsSize) {
        if (insertCount < interleavingFactor && insertCount < itemsToInsertSize) {
            keysInsertionCount = min(itemsToInsertSize - insertCount, BATCH_SIZE);
            topDownInsertion<<<1, BLOCK_SIZE>>>(itemsToInsert + insertCount, keysInsertionCount, d_heapLock, d_partialBuffer, d_Heap, d_mergedResultInsert);
            hipDeviceSynchronize();
            insertCount += keysInsertionCount;
        } 
        if (deleteCount < deletedItemsSize) {
            topDownDeletion<<<1, BLOCK_SIZE>>>(deletedItems + deleteCount, d_heapLock, d_partialBuffer, d_Heap, d_array1, d_array2, d_array3, d_mergedResultDelete);
            hipDeviceSynchronize();
            deleteCount++;
        }
    }

    // Free all allocated memory
    hipFree(d_array1);
    hipFree(d_array2);
    hipFree(d_array3);
    hipFree(d_mergedResultInsert);
    hipFree(d_mergedResultDelete);
}




__host__ void heap_finalise() {
    hipFree(d_partialBuffer);
    hipFree(d_Heap);
    hipFree(d_heapLock);
}
